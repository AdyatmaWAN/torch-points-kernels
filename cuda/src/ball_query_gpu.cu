#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

#define THREADS 1024

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
__global__ void query_ball_point_kernel_dense(int b, int n, int m, float radius,
					      int nsample,
					      const float *__restrict__ new_xyz,
					      const float *__restrict__ xyz,
					      int *__restrict__ idx_out) {

  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  idx_out += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    float new_x = new_xyz[j * 3 + 0];
    float new_y = new_xyz[j * 3 + 1];
    float new_z = new_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      float x = xyz[k * 3 + 0];
      float y = xyz[k * 3 + 1];
      float z = xyz[k * 3 + 2];
      float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx_out[j * nsample + l] = k;
          }
        }
        idx_out[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

__global__ void query_ball_point_kernel_partial_dense(int size_new_xyz,
						      int size_xyz,
						      float radius,
						      int nsample,
						      const float *__restrict__ new_xyz,
						      const float *__restrict__ xyz,
						      const long *__restrict__ batch_new_xyz,
						      const long *__restrict__ batch_xyz,
						      long *__restrict__ idx_out,
						      float * __restrict__ dist_out) {
	// taken from https://github.com/rusty1s/pytorch_cluster/blob/master/cuda/radius_kernel.cu
	const ptrdiff_t batch_idx = blockIdx.x;
	const ptrdiff_t idx = threadIdx.x;

	const ptrdiff_t start_idx_q = batch_new_xyz[batch_idx];
	const ptrdiff_t end_idx_q = batch_new_xyz[batch_idx + 1];

	const ptrdiff_t start_idx_s = batch_xyz[batch_idx];
	const ptrdiff_t end_idx_s = batch_xyz[batch_idx + 1];
	float radius2 = radius * radius;

	for (ptrdiff_t n_q = start_idx_q + idx; n_q < end_idx_q; n_q += THREADS) {
		int count = 0;
		for (ptrdiff_t n_s = start_idx_s; n_s < end_idx_s; n_s++) {
			float dist = 0;
			for (ptrdiff_t d = 0; d < 3; d++) {
				dist += (new_xyz[n_q * 3 + d] - xyz[n_s * 3 + d]) *
					(new_xyz[n_q * 3 + d] - xyz[n_s * 3 + d]);
			}
			if(dist <= radius2){
				if (count == 0){
					for(ptrdiff_t l = 0; l < nsample; ++l){
						idx_out[n_q * nsample + l] = size_xyz;
						dist_out[n_q * nsample + l] = radius2;
					}
				}
				idx_out[n_q*nsample + count] = n_s;
				dist_out[n_q*nsample + count] = dist;
				count++;
		       }
			if(count >= nsample){
				break;
			}
		}
	}
}


void query_ball_point_kernel_dense_wrapper(int b, int n, int m, float radius,
					   int nsample, const float *new_xyz,
					   const float *xyz, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  query_ball_point_kernel_dense<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, nsample, new_xyz, xyz, idx);

  CUDA_CHECK_ERRORS();
}


void query_ball_point_kernel_partial_wrapper(long batch_size,
					     int size_new_xyz,
					     int size_xyz, float radius, int nsample,
					     const float *new_xyz,
					     const float *xyz,
					     const long *batch_new_xyz,
					     const long *batch_xyz,
					     long *idx_out,
					     float *dist_out) {
	query_ball_point_kernel_partial_dense<<<batch_size, THREADS>>>(
		size_new_xyz, size_xyz, radius, nsample, new_xyz, xyz,
		batch_new_xyz, batch_xyz, idx_out, dist_out);

	CUDA_CHECK_ERRORS();
}
